#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* Matrix transpose with Cuda
* Device code.
*/


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <gpumatrix/impl/backend/MatrixOperationInterface.h>

#include "shared_mem.cuh"

#define BLOCK_DIM 16
namespace gpumatrix
{
	namespace impl
	{

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to 
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory 
// so that bank conflicts do not occur when threads address the array column-wise.
template <typename T> __global__ void _row_major_transpose(T *odata, const T *idata, int width, int height)
{
	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

template <typename T> void transpose( T *odata, const T *idata,  int r, int c)  
{									


	dim3 dimGrid(BLOCK_DIM,BLOCK_DIM,1);;
	dim3 dimBlock(int(ceil(float(r)/BLOCK_DIM)),int(ceil(float(c)/BLOCK_DIM)));	

	_row_major_transpose<T><<<dimBlock,dimGrid>>>(odata, idata, r,c);						
}			



template void transpose<double>( double *odata, const double *idata,  int r, int c) ; 
template void transpose<float>( float *odata, const float *idata,  int r, int c)  ;

//
//def _row_wise_sum(tgt, src):
//
//
//    krnl = _get_row_wise_sum_kernel()
//
//    h, w = src.shape
//    assert tgt.shape == (h,)
//
//    threadsize = min(512,int(2** ceil(log(w, 2))));
//
//    gridsize = (h,1);
//    blocksize = (threadsize,1,1)
//
//    sharedsize = threadsize*sizeof(c_float)
//
//    krnl(tgt, src, numpy.int32(h), numpy.int32(w), block = blocksize, grid = gridsize, shared = sharedsize);
//
//// This naive transpose kernel suffers from completely non-coalesced writes.
//// It can be up to 10x slower than the kernel above for large matrices.
//__global__ void transpose_naive(float *odata, float* idata, int width, int height)
//{
//   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
//   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
//   
//   if (xIndex < width && yIndex < height)
//   {
//       unsigned int index_in  = xIndex + width * yIndex;
//       unsigned int index_out = yIndex + height * xIndex;
//       odata[index_out] = idata[index_in]; 
//   }
//}

}
}

